#include "hip/hip_runtime.h"
// Include headers for k-means implementations
#include "thrust_kmeans.cuh"

// Include headers for program setup
#include "argparse.cuh"
#include "io.cuh"
#include "dataset.cuh"
#include "kmeans_kernel.cuh" // For CUDA kernels
#include "error.cuh"         // For HANDLE_CUDA_ERROR


#include <stdio.h>
#include <iostream>

void sequential_kmeans(int num_cluster, KmeansData& data, int max_num_iter, double threshold, bool output_centroids_flag, int seed, bool verbose) {
    if (verbose) {
        std::cout << "Executing Sequential K-Means..." << std::endl;
    }

    // The implementation for the sequential k-means algorithm will go here.
    // For now, it's a placeholder.
    std::cout << "Sequential implementation is not yet complete." << std::endl;
}

void cuda_kmeans(int num_cluster, KmeansData& data, int max_num_iter, double threshold, bool output_centroids_flag, bool verbose) {
    if (verbose) {
        std::cout << "Executing CUDA K-Means..." << std::endl;
    }

    // Extract data dimensions for clarity
    const int num_points = data.num_points;
    const int dims = data.dims;

    // --- 1. Allocate GPU Memory ---
    size_t points_size = (size_t)num_points * dims * sizeof(double);
    size_t centroids_size = (size_t)num_cluster * dims * sizeof(double);
    size_t assignments_size = (size_t)num_points * sizeof(int);

    // Memory for the update step
    double* d_new_centroids_sum;
    int* d_cluster_counts;
    
    // Device pointers
    int* d_cluster_assignments;
    double* d_old_centroids;
    int* d_converged;
    int h_converged = 0; // Host-side convergence flag
    const double threshold_sq = threshold * threshold;

    // Allocate memory for points, centroids, and assignments
    HANDLE_CUDA_ERROR(hipMalloc(&data.d_points, points_size));
    HANDLE_CUDA_ERROR(hipMalloc(&data.d_centroids, centroids_size));
    HANDLE_CUDA_ERROR(hipMalloc(&d_cluster_assignments, assignments_size));
    
    // Allocate memory for update step buffers
    HANDLE_CUDA_ERROR(hipMalloc(&d_new_centroids_sum, centroids_size));
    HANDLE_CUDA_ERROR(hipMalloc(&d_cluster_counts, (size_t)num_cluster * sizeof(int)));

    // Allocate memory for convergence check
    HANDLE_CUDA_ERROR(hipMalloc(&d_old_centroids, centroids_size));
    HANDLE_CUDA_ERROR(hipMalloc(&d_converged, sizeof(int)));

    // --- 2. Copy Initial Data from Host to Device ---
    HANDLE_CUDA_ERROR(hipMemcpy(data.d_points, data.h_points, points_size, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(data.d_centroids, data.h_centroids, centroids_size, hipMemcpyHostToDevice));

    // --- 3. K-Means Iteration Loop ---
    int threads_per_block = 256;
    int point_blocks = (num_points + threads_per_block - 1) / threads_per_block;
    int cluster_blocks = (num_cluster + threads_per_block - 1) / threads_per_block;

    if (verbose) std::cout << "Starting K-Means iterations..." << std::endl;

    for (int iter = 0; iter < max_num_iter; ++iter) {
        // Store current centroids in d_old_centroids to check for convergence
        HANDLE_CUDA_ERROR(hipMemcpy(d_old_centroids, data.d_centroids, centroids_size, hipMemcpyDeviceToDevice));

        // == Assignment Step ==
        // For each point, find the nearest centroid (launches one thread per point)
        assign_clusters_kernel<<<point_blocks, threads_per_block>>>(
            data.d_points, data.d_centroids, d_cluster_assignments, num_points, num_cluster, dims);
        HANDLE_CUDA_ERROR(hipGetLastError());

        // == Update Step ==
        // 1. Reset the summation and count buffers to zero.
        reset_update_buffers_kernel<<<cluster_blocks, threads_per_block>>>(
            d_new_centroids_sum, d_cluster_counts, num_cluster, dims);
        HANDLE_CUDA_ERROR(hipGetLastError());

        // 2. Sum up all the points for each cluster.
        update_centroids_sum_kernel<<<point_blocks, threads_per_block>>>(
            data.d_points, d_cluster_assignments, d_new_centroids_sum, d_cluster_counts, num_points, dims);
        HANDLE_CUDA_ERROR(hipGetLastError());

        // 3. Divide sums by counts to get the new centroids.
        calculate_new_centroids_kernel<<<cluster_blocks, threads_per_block>>>(
            data.d_centroids, d_new_centroids_sum, d_cluster_counts, num_cluster, dims);
        HANDLE_CUDA_ERROR(hipGetLastError());

    }

    if (verbose) std::cout << "K-Means iterations finished." << std::endl;

    // --- 4. Copy Final Centroids from Device to Host ---
    HANDLE_CUDA_ERROR(hipMemcpy(data.h_centroids, data.d_centroids, centroids_size, hipMemcpyDeviceToHost));

    // --- 5. Free GPU Memory ---
    HANDLE_CUDA_ERROR(hipFree(data.d_points));
    HANDLE_CUDA_ERROR(hipFree(data.d_centroids));
    HANDLE_CUDA_ERROR(hipFree(d_cluster_assignments));
    HANDLE_CUDA_ERROR(hipFree(d_old_centroids));
    HANDLE_CUDA_ERROR(hipFree(d_new_centroids_sum));
    HANDLE_CUDA_ERROR(hipFree(d_cluster_counts));
    HANDLE_CUDA_ERROR(hipFree(d_converged));

    // Set device pointers to null to avoid double free issues
    data.d_points = nullptr;
    data.d_centroids = nullptr;
}

void kmeans(int num_cluster, KmeansData& data, int max_num_iter, double threshold, bool output_centroids_flag, int seed, bool verbose, ExecutionMethod method) {
    // Use a switch to dispatch to the correct k-means implementation
    // based on the selected method.
    switch (method) {
        case SEQ:
            sequential_kmeans(num_cluster, data, max_num_iter, threshold, output_centroids_flag, seed, verbose);
            break;
        case CUDA:
           cuda_kmeans(num_cluster, data, max_num_iter, threshold, output_centroids_flag, verbose);
            break;
        case THRUST:
            thrust_kmeans(num_cluster, data, max_num_iter, threshold, output_centroids_flag, seed, verbose);
            break;
        case UNSPECIFIED:
            // This case should ideally not be reached due to argument parsing validation.
            fprintf(stderr, "Error: Execution method is unspecified.\n");
            break;
    }
}

int main(int argc, char* argv[]) {
    KMeansParams params;

    if (!parse_args(argc, argv, params)) {
        return 1; // Exit if argument parsing fails or help is requested
    }

    KmeansData data;
    data.dims = params.dims;

    // Read data from input file into host memory
    if (!read_points(params.inputfilename, data, params.verbose)) {
        return 1; // Exit if data reading fails
    }

    // Initialize the centroids by randomly selecting points from the dataset
    initialize_centroids(data, params.num_cluster, params.seed);

    // Print a sample of the initial centroids if in verbose mode
    // if (params.verbose) data.print_centroids();

    // Call the main k-means logic
    kmeans(params.num_cluster,
           data,               // Pass the KmeansData object
           params.max_num_iter,
           params.threshold,
           params.output_centroids_flag,
           params.seed,
           params.verbose,
           params.method);

    // If requested, print the final centroids
    if (params.output_centroids_flag) {
        // Calculate precision from threshold. e.g., 0.001 -> 3, 0.000001 -> 6
        // A double has about 15-17 decimal digits of precision.
        int precision = 15;
        if (params.threshold > 0) {
            precision = std::max(15, static_cast<int>(ceil(-log10(params.threshold))));
        }
        data.print_centroids(precision);
    }

    // Free all host-side memory.
    delete[] data.h_points;
    delete[] data.h_centroids;

    return 0;
}